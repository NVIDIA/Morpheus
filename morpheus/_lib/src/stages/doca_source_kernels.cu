#include "hip/hip_runtime.h"
// #include "morpheus/doca/doca_context.hpp"

#include "morpheus/doca/common.h"
#include <doca_gpu_device.cuh>
#include <cudf/column/column.hpp>
#include <cudf/column/column_view.hpp>
#include <cudf/strings/detail/utilities.cuh>
#include <cudf/strings/detail/utilities.hpp>
#include <cudf/column/column_factories.hpp>
#include <cudf/column/column_device_view.cuh>
#include <cuda/atomic>
#include <cuda/std/chrono>
#include <memory>
#include <stdio.h>
#include <thrust/iterator/constant_iterator.h>

__device__ char to_hex_16(uint8_t value)
{
    return "0123456789ABCDEF"[value];
}

__device__ int64_t mac_bytes_to_int64(uint8_t* mac)
{
  return static_cast<uint64_t>(mac[0]) << 40
        | static_cast<uint64_t>(mac[1]) << 32
        | static_cast<uint32_t>(mac[2]) << 24
        | static_cast<uint32_t>(mac[3]) << 16
        | static_cast<uint32_t>(mac[4]) << 8
        | static_cast<uint32_t>(mac[5]);
}

__device__ int64_t mac_int64_to_chars(int64_t mac, char* out)
{
  uint8_t mac_0 = (mac >> 40) & (0xFF);
  out[0]  = to_hex_16(mac_0 / 16);
  out[1]  = to_hex_16(mac_0 % 16);
  out[2]  = ':';

  uint8_t mac_1 = (mac >> 32) & (0xFF);
  out[3]  = to_hex_16(mac_1 / 16);
  out[4]  = to_hex_16(mac_1 % 16);
  out[5]  = ':';

  uint8_t mac_2 = (mac >> 24) & (0xFF);
  out[6]  = to_hex_16(mac_2 / 16);
  out[7]  = to_hex_16(mac_2 % 16);
  out[8]  = ':';

  uint8_t mac_3 = (mac >> 16) & (0xFF);
  out[9]  = to_hex_16(mac_3 / 16);
  out[10] = to_hex_16(mac_3 % 16);
  out[11] = ':';

  uint8_t mac_4 = (mac >> 8) & (0xFF);
  out[12] = to_hex_16(mac_4 / 16);
  out[13] = to_hex_16(mac_4 % 16);
  out[14] = ':';

  uint8_t mac_5 = (mac >> 0) & (0xFF);
  out[15] = to_hex_16(mac_5 / 16);
  out[16] = to_hex_16(mac_5 % 16);
}

__global__ void _packet_receive_kernel(
  doca_gpu_rxq_info*                              rxq_info,
  doca_gpu_semaphore_in*                          sem_in,
  uint32_t                                        sem_count,
  uint32_t*                                       sem_idx_begin,
  uint32_t*                                       sem_idx_end,
  uint32_t*                                       packet_count_out,
  uint32_t                                        packet_count_threshold,
  cuda::std::chrono::duration<double>             debounce_threshold,
  cuda::atomic<bool, cuda::thread_scope_system>*  exit_flag
)
{
  // if (threadIdx.x == 0) {
  //   printf("kernel receive: started\n");
  // }

  uint16_t const packet_count_rx_max = 512;
  uint64_t const timeout_ns = 5000000; // 5ms

	__shared__ uint32_t packet_count;

	uintptr_t packet_address;

  if (threadIdx.x == 0){
    *packet_count_out = 0;
  }

  __syncthreads();

  uint32_t sem_idx = *sem_idx_begin;

  auto debounce_checkpoint = cuda::std::chrono::system_clock::now();

  while (*exit_flag == false)
  {
    // ===== WAIT FOR FREE SEM ====================================================================

    __shared__ doca_gpu_semaphore_status sem_status;
    __shared__ bool should_stop_receiving;

    if (*packet_count_out > packet_count_threshold) {
      break;
    }

    if (threadIdx.x == 0)
    {
      should_stop_receiving = false;
      bool first_pass = true;

      while (*exit_flag == false)
      {
        auto ret = doca_gpu_device_semaphore_get_value(
          sem_in + sem_idx,
          &sem_status,
          nullptr,
          nullptr
        );

        if (ret != DOCA_SUCCESS) {
          *exit_flag == true;
          break;
        }

        if (sem_status == DOCA_GPU_SEM_STATUS_FREE) {
          break;
        }

        if (sem_status == DOCA_GPU_SEM_STATUS_READY)
        {
          should_stop_receiving = true;
          break;
        }

        if (first_pass) {
          first_pass = false;
          // printf("kernel receive: waiting on sem %d to become free\n", sem_idx);
        }
      }
    }
    
    __syncthreads();

    if (should_stop_receiving)
    {
      if (threadIdx.x == 0){
        // printf("kernel receive: found sem %d to be ready when expected free.\n", sem_idx);
      }

      break;
    }

    // ===== RECEIVE TO FREE SEM ==================================================================

    DOCA_GPU_VOLATILE(packet_count) = 0;

    auto ret = doca_gpu_device_receive_block(
      rxq_info,
      packet_count_rx_max,
      timeout_ns,
      sem_in + sem_idx,
      true,
      &packet_count,
      &packet_address
    );

    __threadfence();
    __syncthreads();

    if (ret != DOCA_SUCCESS) {
      // printf("kernel receive: setting sem %d to error\n", sem_idx);
      doca_gpu_device_semaphore_update(
        sem_in + sem_idx,
        DOCA_GPU_SEM_STATUS_ERROR,
        packet_count,
        packet_address
      );

      *exit_flag == true;
      continue;
    }

    if (packet_count <= 0)
    {
      auto now = cuda::std::chrono::system_clock::now();

      if (now - debounce_checkpoint > debounce_threshold)
      {
        // if (threadIdx.x == 0)
        // {
        //   printf("kernel receive: timeout while waiting on sem %d to receive packets\n", sem_idx);
        // }
        break;
      }

      continue;
    }

    debounce_checkpoint = cuda::std::chrono::system_clock::now();

    if (threadIdx.x == 0) {
      // printf("kernel receive: setting sem %d to ready\n", sem_idx);
      doca_gpu_device_semaphore_update_status(
        sem_in + sem_idx,
        DOCA_GPU_SEM_STATUS_READY
      );
    }
    __threadfence();
    
    if (threadIdx.x == 0) {
      *packet_count_out = *packet_count_out + packet_count;
      // printf("kernel receive: %d packet(s) received for sem %d\n", packet_count, sem_idx);
    }
    
    sem_idx = (sem_idx + 1) % sem_count;

    if (sem_idx == *sem_idx_end) {
      break;
    }

    __syncthreads();
  }

  if (threadIdx.x == 0)
  {
    *sem_idx_end = sem_idx;
  }

  // if (threadIdx.x == 0)
  // {
  //   printf("kernel receive: end\n");
  // }
}

__global__ void _packet_count_kernel(
  doca_gpu_rxq_info*                                rxq_info,
  doca_gpu_semaphore_in*                            sem_in,
  uint32_t                                          sem_count,
  uint32_t*                                         sem_idx_begin,
  uint32_t*                                         sem_idx_end,
  uint32_t*                                         packet_count_out,
  uint32_t*                                         packets_size_out,
  cuda::atomic<bool, cuda::thread_scope_system>*    exit_flag
)
{
  // if (threadIdx.x == 0) {
  //   printf("kernel count: started\n");
  // }

  *packet_count_out = 0;

  __shared__ doca_gpu_semaphore_status sem_status;
	__shared__ uint32_t  packet_count;

	uintptr_t packet_address;

  auto sem_idx = *sem_idx_begin;

  __syncthreads();

  // ===== WAIT FOR READY SEM ===================================================================

  bool first_pass = true;

  while (*exit_flag == false)
  {
    // get sem info
    auto ret = doca_gpu_device_semaphore_get_value(
      sem_in + sem_idx,
      &sem_status,
      &packet_count,
      &packet_address
    );

    if (ret != DOCA_SUCCESS)
    {
      *exit_flag = true;
      continue;
    }

    __syncthreads();

    if (threadIdx.x == 0 and not first_pass) {
      // printf("kernel count: counting sem %d\n", sem_idx);
    }

    __syncthreads();

    // ===== COUNT PACKETS IN SEM =================================================================

    for (auto packet_idx = 0; packet_idx < packet_count; packet_idx++)
    {
      // compute total packet payload size
      // atomicAdd(packets_size_out, packet_payload_length)
    }

    __syncthreads();

    if (threadIdx.x == 0)
    {
      *packet_count_out += packet_count;

      // TODO: compute packet payload size
      *packets_size_out = 0;
    }

    __syncthreads();

    if (threadIdx.x == 0) {
      // printf("kernel count: setting sem %d to held\n", sem_idx);
      doca_gpu_device_semaphore_update_status(
        sem_in + sem_idx,
        DOCA_GPU_SEM_STATUS_HOLD
      );
    }

    __threadfence();
    __syncthreads();

    // determine if the next sem should be processed

    sem_idx = (sem_idx + 1) % sem_count;

    if (sem_idx == *sem_idx_end)
    {
      break;
    }
  }

  if (threadIdx.x == 0)
  {
    *sem_idx_end = sem_idx;
    // printf("kernel count: %d packets counted for sems [%d, %d) with total payload size %d\n", *packet_count_out, *sem_idx_begin, *sem_idx_end, *packets_size_out);
  }

  // if (threadIdx.x == 0)
  // {
  //   printf("kernel count: done\n");
  // }

  __syncthreads();
}

__global__ void _packet_gather_kernel(
  doca_gpu_rxq_info*                              rxq_info,
  doca_gpu_semaphore_in*                          sem_in,
  uint32_t                                        sem_count,
  uint32_t*                                       sem_idx_begin,
  uint32_t*                                       sem_idx_end,
  uint32_t*                                       packet_count_out,
  uint32_t*                                       packets_size_out,
  uint32_t*                                       packet_length_out,
  int64_t*                                        src_mac_out,
  int64_t*                                        dst_mac_out,
  int64_t*                                        src_ip_out,
  int64_t*                                        dst_ip_out,
  uint16_t*                                       src_port_out,
  uint16_t*                                       dst_port_out,
  cuda::atomic<bool, cuda::thread_scope_system>*  exit_flag
)
{
  // if (threadIdx.x == 0) {
  //   printf("kernel gather: started\n");
  // }

  __shared__ doca_gpu_semaphore_status sem_status;
	__shared__ uint32_t  packet_count;

	uintptr_t packet_address;

  uint32_t sem_idx = *sem_idx_begin;

  // ===== WAIT FOR HELD SEM ======================================================================

  // don't need to wait because we know which sems to process.
  // rule 1: sem at sem_idx_begin must be processed, because we wouldn't be here if there weren't at least one sem to process.
  // rule 2: all sems up to sem_idx_end (exclusive) must be processed.
  // rule 3: if sem_idx_begin == sem_idx_end, sem_idx_begin still gets processed due to rule 1.

  __shared__ uint32_t packet_offset;

  if (threadIdx.x == 0)
  {
    packet_offset = 0;
  }

  __syncthreads();

  while (*exit_flag == false)
  {
    if (threadIdx.x == 0)
    {
      DOCA_GPU_VOLATILE(packet_count) = 0;
    }

    // get sem info
    auto ret = doca_gpu_device_semaphore_get_value(
      sem_in + sem_idx,
      &sem_status,
      &packet_count,
      &packet_address
    );

    if (ret != DOCA_SUCCESS)
    {
      *exit_flag = true;
      continue;
    }

    __syncthreads();

    // copy packets to dataframe

    __shared__ uint32_t stride_start_idx;

    if (threadIdx.x == 0) {
			stride_start_idx = doca_gpu_device_comm_buf_get_stride_idx(
        &(rxq_info->comm_buf),
        packet_address
      );
    }

    __syncthreads();

    for (auto packet_idx = threadIdx.x; packet_idx < packet_count; packet_idx += blockDim.x)
    {
      uint8_t *packet = doca_gpu_device_comm_buf_get_stride_addr(
        &(rxq_info->comm_buf),
        stride_start_idx + packet_idx
      );

      rte_ether_hdr* packet_l2;
      rte_ipv4_hdr*  packet_l3;
      rte_tcp_hdr*   packet_l4;
      uint8_t*       packet_payload;

      get_packet_tcp_headers(
        packet,
        &packet_l2,
        &packet_l3,
        &packet_l4,
        &packet_payload
      );

      auto total_length = BYTE_SWAP16(packet_l3->total_length);

      packet_length_out[total_length] = total_length;

      // mac address printing works
      auto src_mac = packet_l2->s_addr.addr_bytes; // 6 bytes
      auto dst_mac = packet_l2->d_addr.addr_bytes; // 6 bytes

      auto packet_out_idx = packet_offset + packet_idx;

      src_mac_out[packet_out_idx] = mac_bytes_to_int64(src_mac);
      dst_mac_out[packet_out_idx] = mac_bytes_to_int64(src_mac);

      // ip address printing works
      auto src_address  = packet_l3->src_addr;
      auto dst_address  = packet_l3->dst_addr;
      auto src_port     = BYTE_SWAP16(packet_l4->src_port);
      auto dst_port     = BYTE_SWAP16(packet_l4->dst_port);

      // reverse the bytes so int64->ip string kernel works properly.

      auto src_address_rev = (src_address & 0x000000ff) << 24
                           | (src_address & 0x0000ff00) << 8
                           | (src_address & 0x00ff0000) >> 8
                           | (src_address & 0xff000000) >> 24;

      auto dst_address_rev = (dst_address & 0x000000ff) << 24
                           | (dst_address & 0x0000ff00) << 8
                           | (dst_address & 0x00ff0000) >> 8
                           | (dst_address & 0xff000000) >> 24;

      src_ip_out[packet_out_idx] = src_address_rev;
      dst_ip_out[packet_out_idx] = dst_address_rev;

      src_port_out[packet_out_idx] = src_port;
      dst_port_out[packet_out_idx] = dst_port;
    }

    __syncthreads();

    // release sem

    if (threadIdx.x == 0)
    {
      // printf("kernel gather: setting sem %d to free\n", sem_idx);

      doca_gpu_device_semaphore_update_status(
        sem_in + sem_idx,
        DOCA_GPU_SEM_STATUS_FREE
      );

      packet_offset += packet_count;
    }

    __syncthreads();

    // determine if the next sem should be processed

    sem_idx = (sem_idx + 1) % sem_count;

    if (sem_idx == *sem_idx_end)
    {
      break;
    }
  }

  *sem_idx_begin = *sem_idx_end;

  // if (threadIdx.x == 0) {
  //   printf("kernel gather: done\n");
  // }

  __syncthreads();
}

namespace morpheus {
namespace doca {

namespace {

struct integers_to_mac_fn {
  cudf::column_device_view const d_column;
  int32_t const* d_offsets;
  char* d_chars;

  __device__ void operator()(cudf::size_type idx)
  {
    int64_t mac_address = d_column.element<int64_t>(idx);
    char* out_ptr       = d_chars + d_offsets[idx];
    
    mac_int64_to_chars(mac_address, out_ptr);
  }
};

}

std::unique_ptr<cudf::column> integers_to_mac(
  cudf::column_view const& integers,
  rmm::cuda_stream_view stream,
  rmm::mr::device_memory_resource* mr
)
{
  CUDF_EXPECTS(integers.type().id() == cudf::type_id::INT64, "Input column must be type_id::INT64 type");
  CUDF_EXPECTS(integers.null_count() == 0, "integers_to_mac does not support null values.");

  cudf::size_type strings_count = integers.size();

  if (strings_count == 0)
  {
    return cudf::make_empty_column(cudf::type_id::STRING);
  }

  auto offsets_transformer_itr = thrust::constant_iterator<int32_t>(17);
  auto offsets_column = cudf::strings::detail::make_offsets_child_column(
    offsets_transformer_itr,
    offsets_transformer_itr + strings_count,
    stream,
    mr
  );

  auto d_offsets = offsets_column->view().data<int32_t>();

  auto column   = cudf::column_device_view::create(integers, stream);
  auto d_column = *column;

  auto const bytes =
    cudf::detail::get_value<int32_t>(offsets_column->view(), strings_count, stream);

  auto chars_column = cudf::strings::detail::create_chars_child_column(bytes, stream, mr);
  auto d_chars      = chars_column->mutable_view().data<char>();

  thrust::for_each_n(
    rmm::exec_policy(stream),
    thrust::make_counting_iterator<cudf::size_type>(0),
    strings_count,
    integers_to_mac_fn{d_column, d_offsets, d_chars}
  );

  return cudf::make_strings_column(strings_count,
    std::move(offsets_column),
    std::move(chars_column),
    0,
    {});
}

void packet_receive_kernel(
  doca_gpu_rxq_info*                              rxq_info,
  doca_gpu_semaphore_in*                          sem_in,
  uint32_t                                        sem_count,
  uint32_t*                                       sem_idx_begin,
  uint32_t*                                       sem_idx_end,
  uint32_t*                                       packet_count,
  uint32_t                                        packet_count_threshold,
  cuda::std::chrono::duration<double>             debounce_threshold,
  cuda::atomic<bool, cuda::thread_scope_system>*  exit_flag,
  hipStream_t                                    stream
)
{
  _packet_receive_kernel<<<1, 512, 0, stream>>>(
    rxq_info,
    sem_in,
    sem_count,
    sem_idx_begin,
    sem_idx_end,
    packet_count,
    packet_count_threshold,
    debounce_threshold,
    exit_flag
  );
}

void packet_count_kernel(
  doca_gpu_rxq_info*                                rxq_info,
  doca_gpu_semaphore_in*                            sem_in,
  uint32_t                                          sem_count,
  uint32_t*                                         sem_idx_begin,
  uint32_t*                                         sem_idx_end,
  uint32_t*                                         packet_count,
  uint32_t*                                         packets_size,
  cuda::atomic<bool, cuda::thread_scope_system>*    exit_flag,
  hipStream_t                                      stream
)
{
  _packet_count_kernel<<<1, 512, 0, stream>>>(
    rxq_info,
    sem_in,
    sem_count,
    sem_idx_begin,
    sem_idx_end,
    packet_count,
    packets_size,
    exit_flag
  );
}


void packet_gather_kernel(
  doca_gpu_rxq_info*                              rxq_info,
  doca_gpu_semaphore_in*                          sem_in,
  uint32_t                                        sem_count,
  uint32_t*                                       sem_idx_begin,
  uint32_t*                                       sem_idx_end,
  uint32_t*                                       packet_count,
  uint32_t*                                       packets_size,
  uint32_t*                                       packet_length_out,
  int64_t*                                        src_mac_out,
  int64_t*                                        dst_mac_out,
  int64_t*                                        src_ip_out,
  int64_t*                                        dst_ip_out,
  uint16_t*                                       src_port_out,
  uint16_t*                                       dst_port_out,
  cuda::atomic<bool, cuda::thread_scope_system>*  exit_flag,
  hipStream_t                                    stream
)
{
  _packet_gather_kernel<<<1, 512, 0, stream>>>(
    rxq_info,
    sem_in,
    sem_count,
    sem_idx_begin,
    sem_idx_end,
    packet_count,
    packets_size,
    packet_length_out,
    src_mac_out,
    dst_mac_out,
    src_ip_out,
    dst_ip_out,
    src_port_out,
    dst_port_out,
    exit_flag
  );
}

}
}
