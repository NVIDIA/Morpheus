#include "hip/hip_runtime.h"
// #include "morpheus/doca/doca_context.hpp"

#include "morpheus/doca/common.h"
#include <doca_gpu_device.cuh>
#include <cudf/column/column.hpp>
#include <cudf/column/column_view.hpp>
#include <cudf/strings/detail/utilities.cuh>
#include <cudf/strings/detail/utilities.hpp>
#include <cudf/column/column_factories.hpp>
#include <cudf/column/column_device_view.cuh>
#include <cuda/atomic>
#include <cuda/std/chrono>
#include <memory>
#include <stdio.h>
#include <thrust/iterator/constant_iterator.h>
#include <hipcub/hipcub.hpp>

__device__ char to_hex_16(uint8_t value)
{
    return "0123456789ABCDEF"[value];
}

__device__ int64_t mac_bytes_to_int64(uint8_t* mac)
{
  return static_cast<uint64_t>(mac[0]) << 40
        | static_cast<uint64_t>(mac[1]) << 32
        | static_cast<uint32_t>(mac[2]) << 24
        | static_cast<uint32_t>(mac[3]) << 16
        | static_cast<uint32_t>(mac[4]) << 8
        | static_cast<uint32_t>(mac[5]);
}

__device__ int64_t mac_int64_to_chars(int64_t mac, char* out)
{
  uint8_t mac_0 = (mac >> 40) & (0xFF);
  out[0]  = to_hex_16(mac_0 / 16);
  out[1]  = to_hex_16(mac_0 % 16);
  out[2]  = ':';

  uint8_t mac_1 = (mac >> 32) & (0xFF);
  out[3]  = to_hex_16(mac_1 / 16);
  out[4]  = to_hex_16(mac_1 % 16);
  out[5]  = ':';

  uint8_t mac_2 = (mac >> 24) & (0xFF);
  out[6]  = to_hex_16(mac_2 / 16);
  out[7]  = to_hex_16(mac_2 % 16);
  out[8]  = ':';

  uint8_t mac_3 = (mac >> 16) & (0xFF);
  out[9]  = to_hex_16(mac_3 / 16);
  out[10] = to_hex_16(mac_3 % 16);
  out[11] = ':';

  uint8_t mac_4 = (mac >> 8) & (0xFF);
  out[12] = to_hex_16(mac_4 / 16);
  out[13] = to_hex_16(mac_4 % 16);
  out[14] = ':';

  uint8_t mac_5 = (mac >> 0) & (0xFF);
  out[15] = to_hex_16(mac_5 / 16);
  out[16] = to_hex_16(mac_5 % 16);
}

uint32_t const PACKETS_PER_THREAD = 4;
uint32_t const THREADS_PER_BLOCK = 512;
uint32_t const PACKETS_PER_BLOCK = PACKETS_PER_THREAD * THREADS_PER_BLOCK;
// uint32_t const PACKET_RX_TIMEOUT_NS = 5000000; // 5ms
uint32_t const PACKET_RX_TIMEOUT_NS = 50000000; // 50ms

__global__ void _packet_receive_kernel(
  doca_gpu_rxq_info*                              rxq_info,
  doca_gpu_semaphore_in*                          sem_in,
  uint32_t                                        sem_count,
  uint32_t*                                       sem_idx,
  uint32_t*                                       packet_count_out,
  uint32_t*                                       packet_data_size_out
)
{
  if (threadIdx.x == 0)
  {
    *packet_count_out = 0;
    *packet_data_size_out = 0;
  }
  
  __shared__ uint32_t packet_count;
  __shared__ doca_gpu_semaphore_status sem_status;
  
  uintptr_t packet_address;

  if (threadIdx.x == 0)
  {
    while (true)
    {
      auto ret = doca_gpu_device_semaphore_get_value(
        sem_in + *sem_idx,
        &sem_status,
        nullptr,
        nullptr
      );

      if (sem_status == DOCA_GPU_SEM_STATUS_FREE)
      {
        break;
      }
    }
  }

  __syncthreads();

  DOCA_GPU_VOLATILE(packet_count) = 0;

  __syncthreads();

  auto ret = doca_gpu_device_receive_block(
    rxq_info,
    PACKETS_PER_BLOCK,
    PACKET_RX_TIMEOUT_NS,
    nullptr,
    false,
    &packet_count,
    &packet_address
  );

  __threadfence();
  __syncthreads();

  if (packet_count == 0) {
    return;
  }

  __shared__ uint32_t stride_start_idx;

  if (threadIdx.x == 0) {
    *packet_count_out = packet_count;
    stride_start_idx = doca_gpu_device_comm_buf_get_stride_idx(
      &(rxq_info->comm_buf),
      packet_address
    );
  }

  __syncthreads();

  for (auto i = 0; i < PACKETS_PER_THREAD; i++)
  {
    auto packet_idx = threadIdx.x * PACKETS_PER_THREAD + i;

    if (packet_idx >= packet_count) {
      continue;
    }

    uint8_t *packet = doca_gpu_device_comm_buf_get_stride_addr(
      &(rxq_info->comm_buf),
      stride_start_idx + packet_idx
    );

    rte_ether_hdr* packet_l2;
    rte_ipv4_hdr*  packet_l3;
    rte_tcp_hdr*   packet_l4;
    uint8_t*       packet_data;

    get_packet_tcp_headers(
      packet,
      &packet_l2,
      &packet_l3,
      &packet_l4,
      &packet_data
    );

    auto total_length = static_cast<int32_t>(BYTE_SWAP16(packet_l3->total_length));
    auto data_size = total_length - static_cast<int32_t>(packet_l4->dt_off * sizeof(int32_t));

    atomicAdd(packet_data_size_out, data_size);

    // printf("packet_idx(%d) data_size(%d) atom\n", packet_idx, data_size);
  }

  __syncthreads();

  if (threadIdx.x == 0)
  {
    doca_gpu_device_semaphore_update(
      sem_in + *sem_idx,
      DOCA_GPU_SEM_STATUS_HOLD,
      packet_count,
      packet_address
    );
    // doca_gpu_device_semaphore_update_status(
    //   sem_in + *sem_idx,
    //   DOCA_GPU_SEM_STATUS_HOLD
    // );
  }

  __threadfence();
  __syncthreads();
}

__global__ void _packet_gather_kernel(
  doca_gpu_rxq_info*                              rxq_info,
  doca_gpu_semaphore_in*                          sem_in,
  uint32_t                                        sem_count,
  uint32_t*                                       sem_idx,
  uint64_t*                                       timestamp_out,
  int64_t*                                        src_mac_out,
  int64_t*                                        dst_mac_out,
  int64_t*                                        src_ip_out,
  int64_t*                                        dst_ip_out,
  uint16_t*                                       src_port_out,
  uint16_t*                                       dst_port_out,
  int32_t*                                        data_offsets_out,
  char*                                           data_out
)
{
  // Specialize BlockScan for a 1D block of 128 threads of type int
  using BlockScan = hipcub::BlockScan<int32_t, THREADS_PER_BLOCK>;

  // Allocate shared memory for BlockScan
  __shared__ typename BlockScan::TempStorage temp_storage;

  __shared__ doca_gpu_semaphore_status sem_status;
	__shared__ uint32_t packet_count;
  __shared__ uintptr_t packet_address;

  if (threadIdx.x == 0) {

    doca_error_t ret;
    do
    {
      ret = doca_gpu_device_semaphore_get_value_status(
        sem_in + *sem_idx,
        DOCA_GPU_SEM_STATUS_HOLD,
        &sem_status,
        &packet_count,
        &packet_address);

      // auto ret = doca_gpu_device_semaphore_get_value(
      //   sem_in + *sem_idx,
      //   &sem_status,
      //   &packet_count,
      //   &packet_address
      // );

      // if (sem_status == DOCA_GPU_SEM_STATUS_HOLD)
      // {
      //   break;
      // }
    } while(ret == DOCA_ERROR_NOT_FOUND and sem_status != DOCA_GPU_SEM_STATUS_HOLD);
  }

  // __syncthreads();

  // auto ret = doca_gpu_device_semaphore_get_value(
  //   sem_in + *sem_idx,
  //   &sem_status,
  //   &packet_count,
  //   &packet_address
  // );

  __syncthreads();

  __shared__ uint32_t stride_start_idx;

  if (threadIdx.x == 0) {
    stride_start_idx = doca_gpu_device_comm_buf_get_stride_idx(
      &(rxq_info->comm_buf),
      packet_address
    );
  }

  __syncthreads();

  int32_t data_offsets[PACKETS_PER_THREAD];

  for (auto i = 0; i < PACKETS_PER_THREAD; i++)
  {
    auto packet_idx = threadIdx.x * PACKETS_PER_THREAD + i;

    if (packet_idx >= packet_count) {
      continue;
      data_offsets[i] = 0;
    }

    uint8_t *packet = doca_gpu_device_comm_buf_get_stride_addr(
      &(rxq_info->comm_buf),
      stride_start_idx + packet_idx
    );

    rte_ether_hdr* packet_l2;
    rte_ipv4_hdr*  packet_l3;
    rte_tcp_hdr*   packet_l4;
    uint8_t*       packet_data;

    get_packet_tcp_headers(
      packet,
      &packet_l2,
      &packet_l3,
      &packet_l4,
      &packet_data
    );

    auto total_length = static_cast<int32_t>(BYTE_SWAP16(packet_l3->total_length));
    auto data_size = total_length - static_cast<int32_t>(packet_l4->dt_off * sizeof(int32_t));

    data_offsets[i] = data_size;

    // mac address
    auto src_mac = packet_l2->s_addr.addr_bytes; // 6 bytes
    auto dst_mac = packet_l2->d_addr.addr_bytes; // 6 bytes

    src_mac_out[packet_idx] = mac_bytes_to_int64(src_mac);
    dst_mac_out[packet_idx] = mac_bytes_to_int64(dst_mac);

    // ip address
    auto src_address  = packet_l3->src_addr;
    auto dst_address  = packet_l3->dst_addr;

    auto src_address_rev = (src_address & 0x000000ff) << 24
                          | (src_address & 0x0000ff00) << 8
                          | (src_address & 0x00ff0000) >> 8
                          | (src_address & 0xff000000) >> 24;

    auto dst_address_rev = (dst_address & 0x000000ff) << 24
                          | (dst_address & 0x0000ff00) << 8
                          | (dst_address & 0x00ff0000) >> 8
                          | (dst_address & 0xff000000) >> 24;

    src_ip_out[packet_idx] = src_address_rev;
    dst_ip_out[packet_idx] = dst_address_rev;

    // ports
    auto src_port     = BYTE_SWAP16(packet_l4->src_port);
    auto dst_port     = BYTE_SWAP16(packet_l4->dst_port);

    src_port_out[packet_idx] = src_port;
    dst_port_out[packet_idx] = dst_port;
  }

  BlockScan(temp_storage).ExclusiveSum(data_offsets, data_offsets);

  __syncthreads();

  for (auto i = 0; i < PACKETS_PER_THREAD; i++)
  {
    auto packet_idx = threadIdx.x * PACKETS_PER_THREAD + i;

    if (packet_idx >= packet_count) {
      continue;
    }

    uint8_t *packet = doca_gpu_device_comm_buf_get_stride_addr(
      &(rxq_info->comm_buf),
      stride_start_idx + packet_idx
    );

    rte_ether_hdr* packet_l2;
    rte_ipv4_hdr*  packet_l3;
    rte_tcp_hdr*   packet_l4;
    uint8_t*       packet_data;

    get_packet_tcp_headers(
      packet,
      &packet_l2,
      &packet_l3,
      &packet_l4,
      &packet_data
    );

    auto total_length = static_cast<int32_t>(BYTE_SWAP16(packet_l3->total_length));
    auto data_size = total_length - static_cast<int32_t>(packet_l4->dt_off * sizeof(int32_t));

    // printf("packet_idx(%d) data_offset(%d)\n", packet_idx, data_offsets[i]);

    data_offsets_out[packet_idx] = data_offsets[i];

    for (auto data_idx = 0; data_idx < data_size; data_idx++)
    {
      data_out[data_offsets[i] + data_idx] = packet_data[data_idx];
    }
  }

  __syncthreads();

  if (threadIdx.x == 0)
  {
    doca_gpu_device_semaphore_update_status(
      sem_in + *sem_idx,
      DOCA_GPU_SEM_STATUS_FREE
    );
  }

  // // if (threadIdx.x == 0) {
  // //   printf("kernel gather: started\n");
  // // }

  // __shared__ doca_gpu_semaphore_status sem_status;
	// __shared__ uint32_t packet_count;
  // __shared__ uint32_t payload_offset_total;

	// uintptr_t packet_address;

  // uint32_t sem_idx = *sem_idx_begin;

  // // ===== WAIT FOR HELD SEM ======================================================================

  // // don't need to wait because we know which sems to process.
  // // rule 1: sem at sem_idx_begin must be processed, because we wouldn't be here if there weren't at least one sem to process.
  // // rule 2: all sems up to sem_idx_end (exclusive) must be processed.
  // // rule 3: if sem_idx_begin == sem_idx_end, sem_idx_begin still gets processed due to rule 1.

  // __shared__ uint32_t packet_offset;

  // if (threadIdx.x == 0)
  // {
  //   packet_offset = 0;
  //   payload_offset_total = 0;
  // }

  // __syncthreads();

  // while (*exit_flag == false)
  // {
  //   DOCA_GPU_VOLATILE(packet_count) = 0;

  //   __syncthreads();

  //   // get sem info
  //   auto ret = doca_gpu_device_semaphore_get_value(
  //     sem_in + sem_idx,
  //     &sem_status,
  //     &packet_count,
  //     &packet_address
  //   );

  //   if (ret != DOCA_SUCCESS)
  //   {
  //     *exit_flag = true;
  //     continue;
  //   }

  //   __syncthreads();

  //   // copy packets to dataframe

  //   __shared__ uint32_t stride_start_idx;

  //   if (threadIdx.x == 0) {
	// 		stride_start_idx = doca_gpu_device_comm_buf_get_stride_idx(
  //       &(rxq_info->comm_buf),
  //       packet_address
  //     );
  //   }

  //   __syncthreads();

  //   // Obtain a segment of consecutive items that are blocked across threads
  //   uint32_t payload_offsets[PACKETS_PER_THREAD];

  //   for (auto i = 0; i < PACKETS_PER_THREAD; i++)
  //   {
  //     auto packet_idx = threadIdx.x * PACKETS_PER_THREAD + i;

  //     if (packet_idx >= packet_count) {
  //       payload_offsets[i] = 0;
  //       continue;
  //     }

  //     uint8_t *packet = doca_gpu_device_comm_buf_get_stride_addr(
  //       &(rxq_info->comm_buf),
  //       stride_start_idx + packet_idx
  //     );

  //     rte_ether_hdr* packet_l2;
  //     rte_ipv4_hdr*  packet_l3;
  //     rte_tcp_hdr*   packet_l4;
  //     uint8_t*       packet_payload;

  //     get_packet_tcp_headers(
  //       packet,
  //       &packet_l2,
  //       &packet_l3,
  //       &packet_l4,
  //       &packet_payload
  //     );

  //     auto packet_out_idx = packet_offset + packet_idx;

  //     timestamp_out[packet_out_idx] = cuda::std::chrono::duration_cast<cuda::std::chrono::microseconds>(cuda::std::chrono::system_clock::now().time_since_epoch()).count();

  //     auto total_length = BYTE_SWAP16(packet_l3->total_length);
  //     auto payload_size = total_length - (packet_l4->dt_off * sizeof(int));

  //     if (payload_size > 0)
  //     {
  //       printf("payload_size %d\n", payload_size);
  //     }

  //     payload_size_out[packet_out_idx] = payload_size;
  //     payload_offsets[i] = payload_size;

  //     // mac address printing works
  //     auto src_mac = packet_l2->s_addr.addr_bytes; // 6 bytes
  //     auto dst_mac = packet_l2->d_addr.addr_bytes; // 6 bytes

  //     src_mac_out[packet_out_idx] = mac_bytes_to_int64(src_mac);
  //     dst_mac_out[packet_out_idx] = mac_bytes_to_int64(dst_mac);

  //     // ip address printing works
  //     auto src_address  = packet_l3->src_addr;
  //     auto dst_address  = packet_l3->dst_addr;
  //     auto src_port     = BYTE_SWAP16(packet_l4->src_port);
  //     auto dst_port     = BYTE_SWAP16(packet_l4->dst_port);

  //     // reverse the bytes so int64->ip string kernel works properly.

  //     auto src_address_rev = (src_address & 0x000000ff) << 24
  //                          | (src_address & 0x0000ff00) << 8
  //                          | (src_address & 0x00ff0000) >> 8
  //                          | (src_address & 0xff000000) >> 24;

  //     auto dst_address_rev = (dst_address & 0x000000ff) << 24
  //                          | (dst_address & 0x0000ff00) << 8
  //                          | (dst_address & 0x00ff0000) >> 8
  //                          | (dst_address & 0xff000000) >> 24;

  //     src_ip_out[packet_out_idx] = src_address_rev;
  //     dst_ip_out[packet_out_idx] = dst_address_rev;

  //     src_port_out[packet_out_idx] = src_port;
  //     dst_port_out[packet_out_idx] = dst_port;
  //   }

  //   __syncthreads();

  //   uint32_t payload_block_offset;

  //   // Collectively compute the block-wide exclusive prefix sum
  //   BlockScan(temp_storage).ExclusiveSum(payload_offsets, payload_offsets, payload_block_offset);

  //   for (auto i = 0; i < PACKETS_PER_THREAD; i++)
  //   {
  //     auto packet_idx = threadIdx.x * PACKETS_PER_THREAD + i;

  //     if (packet_idx >= packet_count) {
  //       continue;
  //     }

  //     uint8_t *packet = doca_gpu_device_comm_buf_get_stride_addr(
  //       &(rxq_info->comm_buf),
  //       stride_start_idx + packet_idx
  //     );

  //     rte_ether_hdr* packet_l2;
  //     rte_ipv4_hdr*  packet_l3;
  //     rte_tcp_hdr*   packet_l4;
  //     uint8_t*       packet_payload;

  //     get_packet_tcp_headers(
  //       packet,
  //       &packet_l2,
  //       &packet_l3,
  //       &packet_l4,
  //       &packet_payload
  //     );

  //     auto total_length = BYTE_SWAP16(packet_l3->total_length);
  //     auto payload_size = total_length - (packet_l4->dt_off * sizeof(int));

  //     auto payload_offset = payload_offset_total + payload_offsets[i];

  //     for (auto j = 0; j < payload_size; j++)
  //     {
  //       // payload_data_out[payload_offset + j] = packet_payload[j];
  //     }
  //   }

  //   if(threadIdx.x == 0)
  //   {
  //     payload_offset_total += payload_block_offset;
  //   }

  //   // release sem

  //   if (threadIdx.x == 0)
  //   {
  //     // printf("kernel gather: setting sem %d to free\n", sem_idx);

  //     doca_gpu_device_semaphore_update_status(
  //       sem_in + sem_idx,
  //       DOCA_GPU_SEM_STATUS_FREE
  //     );

  //     packet_offset += packet_count;
  //   }

  //   __syncthreads();

  //   // determine if the next sem should be processed

  //   sem_idx = (sem_idx + 1) % sem_count;

  //   if (sem_idx == *sem_idx_end)
  //   {
  //     break;
  //   }
  // }

  // *sem_idx_begin = *sem_idx_end;

  // // if (threadIdx.x == 0) {
  // //   printf("kernel gather: done\n");
  // // }

  // __syncthreads();
}

namespace morpheus {
namespace doca {

namespace {

struct integers_to_mac_fn {
  cudf::column_device_view const d_column;
  int32_t const* d_offsets;
  char* d_chars;

  __device__ void operator()(cudf::size_type idx)
  {
    int64_t mac_address = d_column.element<int64_t>(idx);
    char* out_ptr       = d_chars + d_offsets[idx];
    
    mac_int64_to_chars(mac_address, out_ptr);
  }
};

}

std::unique_ptr<cudf::column> integers_to_mac(
  cudf::column_view const& integers,
  rmm::cuda_stream_view stream,
  rmm::mr::device_memory_resource* mr
)
{
  CUDF_EXPECTS(integers.type().id() == cudf::type_id::INT64, "Input column must be type_id::INT64 type");
  CUDF_EXPECTS(integers.null_count() == 0, "integers_to_mac does not support null values.");

  cudf::size_type strings_count = integers.size();

  if (strings_count == 0)
  {
    return cudf::make_empty_column(cudf::type_id::STRING);
  }

  auto offsets_transformer_itr = thrust::constant_iterator<int32_t>(17);
  auto offsets_column = cudf::strings::detail::make_offsets_child_column(
    offsets_transformer_itr,
    offsets_transformer_itr + strings_count,
    stream,
    mr
  );

  auto d_offsets = offsets_column->view().data<int32_t>();

  auto column   = cudf::column_device_view::create(integers, stream);
  auto d_column = *column;

  auto const bytes =
    cudf::detail::get_value<int32_t>(offsets_column->view(), strings_count, stream);

  auto chars_column = cudf::strings::detail::create_chars_child_column(bytes, stream, mr);
  auto d_chars      = chars_column->mutable_view().data<char>();

  thrust::for_each_n(
    rmm::exec_policy(stream),
    thrust::make_counting_iterator<cudf::size_type>(0),
    strings_count,
    integers_to_mac_fn{d_column, d_offsets, d_chars}
  );

  return cudf::make_strings_column(strings_count,
    std::move(offsets_column),
    std::move(chars_column),
    0,
    {});
}

struct picker {
  uint32_t* lengths;
  __device__ uint32_t operator()(cudf::size_type idx){
    if (lengths[idx] > 0)
    {
      printf("pdl: %d\n", lengths[idx]);
    }
    return lengths[idx];
  }
};

std::unique_ptr<cudf::column> packet_data_to_column(
  cudf::size_type packet_count,
  rmm::device_uvector<char> && packet_data_chars,
  rmm::device_uvector<uint32_t> && packet_data_lengths,
  rmm::cuda_stream_view stream,
  rmm::mr::device_memory_resource* mr)
{
  auto offsets_transformer_itr = thrust::make_transform_iterator(
    thrust::make_counting_iterator<int32_t>(0),
    picker{packet_data_lengths.data()}
    // [data_lengths = packet_data_lengths.data()] __device__(cudf::size_type idx) {
    //   return data_lengths[idx];
    // }
  );

  auto payload_offsets_column = cudf::strings::detail::make_offsets_child_column(
    offsets_transformer_itr,
    offsets_transformer_itr + packet_count,
    stream,
    mr
  );

  stream.synchronize();

  auto packet_data_chars_size = packet_data_chars.size();
  auto packet_data_chars_col  = std::make_unique<cudf::column>(
    cudf::data_type{cudf::type_to_id<char>()},
    packet_data_chars_size,
    packet_data_chars.release());

  uint32_t last_offset;

  hipMemcpy(&last_offset, payload_offsets_column->view().data<uint32_t>() + packet_count - 1, sizeof(uint32_t), hipMemcpyDeviceToHost);

  std::cout << "last offset: " << last_offset << " "
            << "chars size: " << packet_data_chars_size
            << std::endl;

  return cudf::make_strings_column(
    packet_count,
    std::move(payload_offsets_column),
    std::move(packet_data_chars_col),
    0,
    {}
  );
}

void packet_receive_kernel(
  doca_gpu_rxq_info*                              rxq_info,
  doca_gpu_semaphore_in*                          sem_in,
  uint32_t                                        sem_count,
  uint32_t*                                       sem_idx,
  uint32_t*                                       packet_count,
  uint32_t*                                       packet_data_size,
  hipStream_t                                    stream
)
{
  _packet_receive_kernel<<<1, THREADS_PER_BLOCK, 0, stream>>>(
    rxq_info,
    sem_in,
    sem_count,
    sem_idx,
    packet_count,
    packet_data_size
  );
}

void packet_gather_kernel(
  doca_gpu_rxq_info*                              rxq_info,
  doca_gpu_semaphore_in*                          sem_in,
  uint32_t                                        sem_count,
  uint32_t*                                       sem_idx,
  uint64_t*                                       timestamp_out,
  int64_t*                                        src_mac_out,
  int64_t*                                        dst_mac_out,
  int64_t*                                        src_ip_out,
  int64_t*                                        dst_ip_out,
  uint16_t*                                       src_port_out,
  uint16_t*                                       dst_port_out,
  int32_t*                                        data_offsets_out,
  char*                                           data_out,
  hipStream_t                                    stream
)
{
  _packet_gather_kernel<<<1, THREADS_PER_BLOCK, 0, stream>>>(
    rxq_info,
    sem_in,
    sem_count,
    sem_idx,
    timestamp_out,
    src_mac_out,
    dst_mac_out,
    src_ip_out,
    dst_ip_out,
    src_port_out,
    dst_port_out,
    data_offsets_out,
    data_out
  );
}

}
}
