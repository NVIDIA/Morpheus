#include "hip/hip_runtime.h"
// #include "morpheus/doca/doca_context.hpp"

#include "morpheus/doca/common.h"
#include <doca_gpu_device.cuh>
#include <stdio.h>

// __device__ void
// get_packet_tcp_headers(
//   uint8_t* pkt,
//   rte_ether_hdr **l2_hdr,
//   rte_ipv4_hdr **l3_hdr,
//   rte_tcp_hdr **l4_hdr,
//   uint8_t **l5_pld
// )
// {
// 	*l2_hdr = (rte_ether_hdr *) pkt;
// 	*l3_hdr = (rte_ipv4_hdr *)  (((uint8_t*) (*l2_hdr)) + RTE_ETHER_HDR_LEN);
// 	*l4_hdr = (rte_tcp_hdr *)   (((uint8_t*) (*l3_hdr)) + (uint8_t)(((*l3_hdr)->version_ihl & RTE_IPV4_HDR_IHL_MASK) * RTE_IPV4_IHL_MULTIPLIER));
// 	*l5_pld = ((uint8_t*)(*l4_hdr)) + (uint8_t)((*l4_hdr)->dt_off * sizeof(int));
// }

__global__ void _kernel_receive_persistent(
  doca_gpu_rxq_info*      rxq_info,
  doca_gpu_semaphore_in*  sem_in,
  uint32_t                sem_count
)
{
  if (blockIdx.x != 0) {
    return;
  }

  if (threadIdx.x == 0) {
    printf("kernel: Hello World!\n");
  }

  __syncthreads();

	__shared__ uint32_t  packet_count;
	__shared__ uintptr_t packet_address;

  for (auto sem_idx = 0; sem_idx < sem_count;)
  {
    uint16_t packet_count_rx_max = 2048;
    uint64_t timeout_ns = 5000000; // 5ms

    auto ret = doca_gpu_device_receive_block(
      rxq_info,
      packet_count_rx_max,
      timeout_ns,
      sem_in + sem_idx,
      true,
      &packet_count,
      &packet_address
    );

		__threadfence();
		__syncthreads();

    if (ret != DOCA_SUCCESS) {
      if (threadIdx.x == 0) {
        printf("kernel: packet receive failed.\n");
      }

      __syncthreads();

      // TODO: determine if this should be called on all threads or not.
			doca_gpu_device_semaphore_update(
        &(sem_in[sem_idx]),
        DOCA_GPU_SEM_STATUS_ERROR,
        packet_count,
        packet_address
      );
    }

    __syncthreads();

    auto should_exit = __any_sync(0xffffffff, ret != DOCA_SUCCESS);

    if (should_exit) {
      if (threadIdx.x == 0) {
        printf("kernel: exiting due to error.\n");
      }

      __syncthreads();
      return;
    }

    __syncthreads();

    // if (packet_count > 0) {
    //   if (threadIdx.x == 0) {
    //     printf("kernel: %d packet(s) recieved.\n", packet_count);
    //   }
    //   sem_idx++;
    // }

    __syncthreads();

    __shared__ uint32_t stride_start_idx;

    if (threadIdx.x == 0) {
			stride_start_idx = doca_gpu_device_comm_buf_get_stride_idx(
        &rxq_info->comm_buf,
        packet_address
      );
    }

    __syncthreads();

    for (auto packet_idx = threadIdx.x; packet_idx < packet_count; packet_idx += blockDim.x)
    {
      uint8_t* packet = doca_gpu_device_comm_buf_get_stride_addr(
        &rxq_info->comm_buf,
        stride_start_idx + packet_idx
      );

      rte_ether_hdr* packet_l2;
      rte_ipv4_hdr*  packet_l3;
      rte_tcp_hdr*   packet_l4;
      uint8_t*       packet_payload;

      get_packet_tcp_headers(
        packet,
        &packet_l2,
        &packet_l3,
        &packet_l4,
        &packet_payload
      );

      uint32_t tmp =
        ((uint16_t*)packet_l3)[0] + ((uint16_t*)packet_l3)[1] + ((uint16_t*)packet_l3)[2] +
			  ((uint16_t*)packet_l3)[3] + ((uint16_t*)packet_l3)[4] + //((uint16_t*)packet_l3)[5] +
			  ((uint16_t*)packet_l3)[6] + ((uint16_t*)packet_l3)[7] + ((uint16_t*)packet_l3)[8] +
			  ((uint16_t*)packet_l3)[9];

      uint16_t checksum = ~((uint16_t)(tmp & 0xFFFF) + (uint16_t)(tmp >> 16));

      if (packet_l3->hdr_checksum != checksum) {
        printf("checksum mismatch: %d / %d\n", packet_l3->hdr_checksum, checksum);
      }

      if (packet_payload[0] == 'H')
      {
        printf(
          "IP: %x / %x len: %d, TCP: %d / %d dtoff: %x ih3 len: %d, pld: %x %x %x %x\n",
          packet_l3->src_addr,
          packet_l3->dst_addr,
          BYTE_SWAP16(packet_l3->total_length),
          BYTE_SWAP16(packet_l4->src_port),
          BYTE_SWAP16(packet_l4->dst_port),
          packet_l4->dt_off,
          (uint8_t)((packet_l3->version_ihl & RTE_IPV4_HDR_IHL_MASK) * RTE_IPV4_IHL_MULTIPLIER),
          packet_payload[0],
          packet_payload[1],
          packet_payload[2],
          packet_payload[3]
        );
      }

      __syncthreads();
    }

    if (threadIdx.x == 0) {
			doca_gpu_device_semaphore_update_status(
        &(sem_in[sem_idx]),
        DOCA_GPU_SEM_STATUS_DONE
      );
    }

    __syncthreads();
  }

}

void doca_receive_persistent(
  doca_gpu_rxq_info*      rxq_info,
  doca_gpu_semaphore_in*  sem_in,
  uint32_t                sem_count,
  hipStream_t stream
)
{
  _kernel_receive_persistent<<<1, 512, 0, stream>>>(
    rxq_info,
    sem_in,
    sem_count
  );
}
