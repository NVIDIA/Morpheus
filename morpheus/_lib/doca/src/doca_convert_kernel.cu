#include "hip/hip_runtime.h"
/*
 * SPDX-FileCopyrightText: Copyright (c) 2022-2024, NVIDIA CORPORATION & AFFILIATES. All rights reserved.
 * SPDX-License-Identifier: Apache-2.0
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 * http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "morpheus/doca/common.hpp"
#include "morpheus/doca/packets.hpp"
#include "morpheus/utilities/error.hpp"

#include <hipcub/hipcub.hpp>
#include <cuda/std/chrono>
#include <doca_eth_rxq.h>
#include <doca_gpunetio.h>
#include <doca_gpunetio_dev_buf.cuh>
#include <doca_gpunetio_dev_eth_rxq.cuh>
#include <doca_gpunetio_dev_sem.cuh>
#include <matx.h>
#include <rmm/device_buffer.hpp>
#include <rmm/exec_policy.hpp>
#include <rte_ether.h>
#include <rte_ip.h>
#include <rte_tcp.h>
#include <stdio.h>
#include <thrust/gather.h>
#include <thrust/iterator/constant_iterator.h>
#include <thrust/iterator/counting_iterator.h>
#include <memory>

__global__ void _packet_gather_payload_kernel(
  int32_t  packet_count,
  uintptr_t*  packets_buffer,
  uint32_t* header_sizes,
  uint32_t* payload_sizes,
  uint8_t*  payload_chars_out
)
{
  int pkt_idx = threadIdx.x;
  int j = 0;

  while (pkt_idx < packet_count) {
    uint8_t* pkt_hdr_addr = (uint8_t*)(packets_buffer[pkt_idx] + header_sizes[pkt_idx]);
    // if (!pkt_hdr_addr)
    //   continue;
    for (j = 0; j < payload_sizes[pkt_idx]; j++)
      payload_chars_out[(MAX_PKT_SIZE * pkt_idx) + j] = pkt_hdr_addr[j];
    for (; j < MAX_PKT_SIZE; j++)
      payload_chars_out[(MAX_PKT_SIZE * pkt_idx) + j] = '\0';
    pkt_idx += blockDim.x;
  }

#if 0

  // Specialize BlockScan for a 1D block of 128 threads of type int
  using BlockScan = hipcub::BlockScan<int32_t, THREADS_PER_BLOCK>;
  // Allocate shared memory for BlockScan
  __shared__ typename BlockScan::TempStorage temp_storage;
  int32_t payload_offsets[PACKETS_PER_THREAD];
  /* Th0 will work on first 4 packets, etc.. */
  for (auto i = 0; i < PACKETS_PER_THREAD; i++) {
    auto packet_idx = threadIdx.x * PACKETS_PER_THREAD + i;
    if (packet_idx >= packet_count)
      payload_offsets[i] = 0;
    else
      payload_offsets[i] = payload_sizes[packet_idx];
  }
  __syncthreads();

  /* Calculate the right payload offset for each thread */
  int32_t data_offsets_agg;
  BlockScan(temp_storage).ExclusiveSum(payload_offsets, payload_offsets, data_offsets_agg);
  __syncthreads();

  for (auto i = 0; i < PACKETS_PER_THREAD; i++) {
    auto packet_idx = threadIdx.x * PACKETS_PER_THREAD + i;
    if (packet_idx >= packet_count)
      continue;

    auto payload_size = payload_sizes[packet_idx];
    for (auto j = 0; j < payload_size; j++) {
      auto value = *(((uint8_t*)packets_buffer[packet_idx]) + header_sizes[packet_idx] + j);
      payload_chars_out[payload_offsets[i] + j] = value;
      // printf("payload %d size %d : 0x%1x / 0x%1x addr %lx\n",
      //     payload_offsets[i] + j, payload_size,
      //     payload_chars_out[payload_offsets[i] + j], value,
      //     packets_buffer[packet_idx]);
    }
  }
#endif
}

__global__ void _packet_gather_header_kernel(
  int32_t   packet_count,
  uintptr_t*  packets_buffer,
  uint32_t* header_sizes,
  uint32_t* payload_sizes,
  uint8_t*  header_src_ip_addr
)
{
  int pkt_idx = threadIdx.x;

  while (pkt_idx < packet_count) {
    uint8_t* pkt_hdr_addr = (uint8_t*)(packets_buffer[pkt_idx]);
    // if (!pkt_hdr_addr)
    //   continue;
    int len = ip_to_string(((struct eth_ip *)pkt_hdr_addr)->l3_hdr.src_addr, header_src_ip_addr + (IP_ADDR_STRING_LEN * pkt_idx));
    while (len < IP_ADDR_STRING_LEN)
      header_src_ip_addr[(IP_ADDR_STRING_LEN * pkt_idx) + len++] = '\0';
    pkt_idx += blockDim.x;
  }
}

namespace morpheus {
namespace doca {

std::pair<uint32_t, uint32_t> gather_sizes(
    int32_t packet_count,
    uint32_t* fixed_header_size_list,
    uint32_t* fixed_payload_size_list,
    rmm::cuda_stream_view stream
)
{
    auto header_sizes_tensor = matx::make_tensor<uint32_t>(fixed_header_size_list, {packet_count});
    auto payload_sizes_tensor = matx::make_tensor<uint32_t>(fixed_payload_size_list, {packet_count});

    auto header_bytes_tensor = matx::make_tensor<uint32_t>({1});
    auto payload_bytes_tensor = matx::make_tensor<uint32_t>({1});

    (header_bytes_tensor = matx::sum(header_sizes_tensor)).run(stream.value());
    (payload_bytes_tensor = matx::sum(payload_sizes_tensor)).run(stream.value());

    hipStreamSynchronize(stream);
    return {header_bytes_tensor(0), payload_bytes_tensor(0)};
}

rmm::device_buffer sizes_to_offsets(
    int32_t packet_count,
    uint32_t* sizes_buff,
    rmm::cuda_stream_view stream)
{
    // The cudf offsets column wants int32
    const auto out_elem_count = packet_count+1;
    const auto out_byte_size = out_elem_count*sizeof(int32_t);
    rmm::device_buffer out_buffer(out_byte_size, stream);

    auto sizes_tensor = matx::make_tensor<uint32_t>(sizes_buff, {packet_count});
    auto cum_tensor = matx::make_tensor<int32_t>({packet_count});

    // first element needs to be a 0
    auto zero_tensor = matx::make_tensor<int32_t>({1});
    zero_tensor.SetVals({0});

    auto offsets_tensor = matx::make_tensor<int32_t>(static_cast<int32_t*>(out_buffer.data()), {out_elem_count});


    (cum_tensor = matx::cumsum(matx::as_type<int32_t>(sizes_tensor))).run(stream.value());
    (offsets_tensor = matx::concat(0, zero_tensor, cum_tensor)).run(stream.value());

    hipStreamSynchronize(stream);

    return out_buffer;
}

void gather_payload(
  int32_t      packet_count,
  uintptr_t*   packets_buffer,
  uint32_t*    header_sizes,
  uint32_t*    payload_sizes,
  uint8_t*     dst_buff,
  rmm::cuda_stream_view stream,
  rmm::mr::device_memory_resource* mr)
{
  _packet_gather_payload_kernel<<<1, THREADS_PER_BLOCK, 0, stream>>>(
    packet_count,
    packets_buffer,
    header_sizes,
    payload_sizes,
    dst_buff
  );

}

void gather_header(
  int32_t      packet_count,
  uintptr_t*   packets_buffer,
  uint32_t*    header_sizes,
  uint32_t*    payload_sizes,
  uint8_t*     dst_buff,
  rmm::cuda_stream_view stream,
  rmm::mr::device_memory_resource* mr)
{

  _packet_gather_header_kernel<<<1, THREADS_PER_BLOCK, 0, stream>>>(
    packet_count,
    packets_buffer,
    header_sizes,
    payload_sizes,
    dst_buff
  );
}


} //doca
} //morpheus
